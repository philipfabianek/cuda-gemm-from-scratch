#include <cstdio>
#include <vector>

#include "kernels/00_cublas.cuh"
#include "kernels/01_naive.cuh"
#include "utils.cuh"

void run_kernel(int kernel_id, hipblasHandle_t handle, int M, int N, int K,
                float alpha, float *d_A, float *d_B, float beta, float *d_C) {
  switch (kernel_id) {
  case 0:
    run_cublas_kernel(handle, M, K, N, alpha, d_A, d_B, beta, d_C);
    break;
  case 1:
    run_naive_kernel(M, N, K, alpha, d_A, d_B, beta, d_C);
    break;
  default:
    fprintf(stderr, "Error: Invalid kernel ID.\n");
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char **argv) {
  if (argc != 2) {
    fprintf(stderr, "Usage: %s <kernel_id>\n", argv[0]);
    return 1;
  }
  int kernel_id = std::stoi(argv[1]);

  // Benchmark configuration
  bool random_initialization = true;
  int size = 2048;
  int M = size;
  int N = size;
  int K = size;
  float alpha = 2.0f;
  float beta = 0.5f;

  // Create cublas handle
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Prepare host matrices
  std::vector<float> h_A(M * K);
  std::vector<float> h_B(K * N);
  std::vector<float> h_C(M * N);

  initialize_matrix(h_A, M, K, random_initialization);
  initialize_matrix(h_B, K, N, random_initialization);
  initialize_matrix(h_C, M, N, random_initialization);

  // Store initial C matrix for verification
  std::vector<float> h_C_initial = h_C;

  // Prepare device variables
  float *d_A, *d_B, *d_C, *d_C_reference;
  size_t a_size = h_A.size() * sizeof(float);
  size_t b_size = h_B.size() * sizeof(float);
  size_t c_size = h_C.size() * sizeof(float);

  CUDA_CHECK(hipMalloc(&d_A, a_size));
  CUDA_CHECK(hipMalloc(&d_B, b_size));
  CUDA_CHECK(hipMalloc(&d_C, c_size));
  CUDA_CHECK(hipMalloc(&d_C_reference, c_size));

  CUDA_CHECK(hipMemcpy(d_A, h_A.data(), a_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, h_B.data(), b_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_C, h_C.data(), c_size, hipMemcpyHostToDevice));

  // Generate cublas reference, reset d_C afterwards
  CUDA_CHECK(
      hipMemcpy(d_C_reference, h_C.data(), c_size, hipMemcpyHostToDevice));
  run_cublas_kernel(handle, M, K, N, alpha, d_A, d_B, beta, d_C_reference);
  CUDA_CHECK(hipMemcpy(d_C, h_C.data(), c_size, hipMemcpyHostToDevice));

  // Warm-up run, reset d_C afterwards
  run_kernel(kernel_id, handle, M, N, K, alpha, d_A, d_B, beta, d_C);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipMemcpy(d_C, h_C.data(), c_size, hipMemcpyHostToDevice));

  // Create events for timing
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  // Execute the kernel, no need to check errors now
  run_kernel(kernel_id, handle, M, N, K, alpha, d_A, d_B, beta, d_C);

  // Record the end time and synchronize
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  // Calculate elapsed time
  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
  printf("Kernel: Naive\n");
  printf("Matrix Size: %dx%d and %dx%d\n", M, K, K, N);
  printf("Execution time: %f ms\n", milliseconds);

  // Calculate TFLOPS
  long long total_ops = (long long)2 * M * K * N;
  double tflops = (double)total_ops / (milliseconds / 1000.0) / 1e12;
  printf("Performance: %.2f TFLOPS\n", tflops);

  // Verify results
  verify_with_cublas_reference(M, N, d_C, d_C_reference);

  // Free memory and destroy events
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));
  CUDA_CHECK(hipFree(d_C_reference));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  hipblasDestroy(handle);

  return 0;
}