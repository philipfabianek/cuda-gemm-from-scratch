#include <cstdio>
#include <vector>

#include "arg_parser.cuh"
#include "kernels/00_cublas.cuh"
#include "kernels/01_naive.cuh"
#include "kernels/02_tiled.cuh"
#include "kernels/03_1D_coarsened.cuh"
#include "kernels/04_2D_coarsened.cuh"
#include "utils.cuh"

void run_kernel(int kernel_id, hipblasHandle_t handle, int M, int N, int K,
                float alpha, const float *d_A, const float *d_B, float beta,
                float *d_C) {
  switch (kernel_id) {
  case 0:
    run_cublas_kernel(handle, M, N, K, alpha, d_A, d_B, beta, d_C);
    break;
  case 1:
    run_naive_kernel(M, N, K, alpha, d_A, d_B, beta, d_C);
    break;
  case 2:
    run_tiled_kernel(M, N, K, alpha, d_A, d_B, beta, d_C);
    break;
  case 3:
    run_1D_coarsened_kernel(M, N, K, alpha, d_A, d_B, beta, d_C);
    break;
  case 4:
    run_2D_coarsened_kernel(M, N, K, alpha, d_A, d_B, beta, d_C);
    break;
  default:
    fprintf(stderr, "Error: Invalid kernel ID.\n");
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char **argv) {
  ArgParser parser(argc, argv);

  if (parser.cmd_option_exists("--help")) {
    printf("Usage: %s --kernel <id> [--size N] [--repeats N]\n", argv[0]);
    return 0;
  }

  // Make kernel_id a named, required argument
  if (!parser.cmd_option_exists("--kernel")) {
    fprintf(stderr, "Error: Missing required argument --kernel <id>\n");
    printf("Usage: %s --kernel <id> [--size N] [--repeats N]\n", argv[0]);
    return 1;
  }

  // Load CLI arguments
  int kernel_id = parser.get_cmd_option<int>("--kernel", 1);
  int size = parser.get_cmd_option<int>("--size", 1024);
  int repeats = parser.get_cmd_option<int>("--repeats", 100);

  // Benchmark configuration
  int M = size;
  int N = size;
  int K = size;
  float alpha = 2.0f;
  float beta = 0.5f;

  // Create cublas handle
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Prepare host matrices
  std::vector<float> h_A(M * K);
  std::vector<float> h_B(K * N);
  std::vector<float> h_C(M * N);

  initialize_matrix(h_A, M, K);
  initialize_matrix(h_B, K, N);
  initialize_matrix(h_C, M, N);

  // Store initial C matrix for verification
  std::vector<float> h_C_initial = h_C;

  // Prepare device variables
  float *d_A, *d_B, *d_C, *d_C_reference;
  size_t a_size = h_A.size() * sizeof(float);
  size_t b_size = h_B.size() * sizeof(float);
  size_t c_size = h_C.size() * sizeof(float);

  CUDA_CHECK(hipMalloc(&d_A, a_size));
  CUDA_CHECK(hipMalloc(&d_B, b_size));
  CUDA_CHECK(hipMalloc(&d_C, c_size));
  CUDA_CHECK(hipMalloc(&d_C_reference, c_size));

  CUDA_CHECK(hipMemcpy(d_A, h_A.data(), a_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, h_B.data(), b_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_C, h_C.data(), c_size, hipMemcpyHostToDevice));

  // Generate cublas reference, reset d_C afterwards
  CUDA_CHECK(
      hipMemcpy(d_C_reference, h_C.data(), c_size, hipMemcpyHostToDevice));
  run_cublas_kernel(handle, M, N, K, alpha, d_A, d_B, beta, d_C_reference);
  CUDA_CHECK(hipMemcpy(d_C, h_C.data(), c_size, hipMemcpyHostToDevice));

  // Warm-up run, check for errors, verify results and reset d_C afterwards
  run_kernel(kernel_id, handle, M, N, K, alpha, d_A, d_B, beta, d_C);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  verify_with_cublas_reference(M, N, d_C, d_C_reference);
  CUDA_CHECK(hipMemcpy(d_C, h_C.data(), c_size, hipMemcpyHostToDevice));

  // Create events for timing
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  // Execute the kernel, no need to check errors here
  for (int i = 0; i < repeats; ++i) {
    run_kernel(kernel_id, handle, M, N, K, alpha, d_A, d_B, beta, d_C);
  }

  // Record the end time and synchronize
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  // Measure performance
  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
  float avg_milliseconds = milliseconds / repeats;
  long long total_ops = (long long)2 * M * K * N;
  double gflops = (double)total_ops / (avg_milliseconds / 1000.0) / 1e9;
  printf("Kernel ID %d - Average time: (%f) ms, performance: (%.2f) GFLOPS, "
         "size: (%d).\n",
         kernel_id, avg_milliseconds, gflops, size);

  // Free memory and destroy events
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));
  CUDA_CHECK(hipFree(d_C_reference));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  hipblasDestroy(handle);

  return 0;
}